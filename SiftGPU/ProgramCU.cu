
#include <hip/hip_runtime.h>
////////////////////////////////////////////////////////////////////////////
//	File:		ProgramCU.cu
//	Author:		Changchang Wu
//	Description : implementation of ProgramCU and all CUDA kernels
//
//	Copyright (c) 2007 University of North Carolina at Chapel Hill
//	All Rights Reserved
//
//	Permission to use, copy, modify and distribute this software and its
//	documentation for educational, research and non-profit purposes, without
//	fee, and without a written agreement is hereby granted, provided that the
//	above copyright notice and the following paragraph appear in all copies.
//	
//	The University of North Carolina at Chapel Hill make no representations
//	about the suitability of this software for any purpose. It is provided
//	'as is' without express or implied warranty. 
//
//	Please send BUG REPORTS to ccwu@cs.unc.edu
//
////////////////////////////////////////////////////////////////////////////

#if defined(CUDA_SIFTGPU_ENABLED)

#include "GL/glew.h"
#include "stdio.h"

#include "CuTexImage.h"
#include "ProgramCU.h"
#include "GlobalUtil.h"

//----------------------------------------------------------------
//Begin SiftGPU setting section.
//////////////////////////////////////////////////////////
#define IMUL(X,Y) __mul24(X,Y)
//#define FDIV(X,Y) ((X)/(Y))
#define FDIV(X,Y) __fdividef(X,Y)

/////////////////////////////////////////////////////////
//filter kernel width range (don't change this)
#define KERNEL_MAX_WIDTH 33
#define KERNEL_MIN_WIDTH 5

//////////////////////////////////////////////////////////
//horizontal filter block size (32, 64, 128, 256, 512)
#define FILTERH_TILE_WIDTH 128
//#define FILTERH_TILE_WIDTH 256
//#define FILTERH_TILE_WIDTH 160
//thread block for vertical filter. FILTERV_BLOCK_WIDTH can be (4, 8 or 16)
#define FILTERV_BLOCK_WIDTH 16
#define FILTERV_BLOCK_HEIGHT 32
//The corresponding image patch for a thread block
#define FILTERV_PIXEL_PER_THREAD 4
#define FILTERV_TILE_WIDTH FILTERV_BLOCK_WIDTH
#define FILTERV_TILE_HEIGHT (FILTERV_PIXEL_PER_THREAD * FILTERV_BLOCK_HEIGHT)


//////////////////////////////////////////////////////////
//thread block size for computing Difference of Gaussian
#define DOG_BLOCK_LOG_DIMX 7
#define DOG_BLOCK_LOG_DIMY 0
#define DOG_BLOCK_DIMX (1 << DOG_BLOCK_LOG_DIMX)
#define DOG_BLOCK_DIMY (1 << DOG_BLOCK_LOG_DIMY)

//////////////////////////////////////////////////////////
//thread block size for keypoint detection
#define KEY_BLOCK_LOG_DIMX 3
#define KEY_BLOCK_LOG_DIMY 3
#define KEY_BLOCK_DIMX (1<<KEY_BLOCK_LOG_DIMX)
#define KEY_BLOCK_DIMY (1<<KEY_BLOCK_LOG_DIMY)
//#define KEY_OFFSET_ONE
//make KEY_BLOCK_LOG_DIMX 4 will make the write coalesced..
//but it seems uncoalesced writes don't affect the speed

//////////////////////////////////////////////////////////
//thread block size for initializing list generation (64, 128, 256, 512 ...) 实例化列表生成的线程块大小！
#define HIST_INIT_WIDTH 128
//thread block size for generating feature list (32, 64, 128, 256, 512, ...)生成特征列表的线程块大小
#define LISTGEN_BLOCK_DIM 128


/////////////////////////////////////////////////////////
//how many keypoint orientations to compute in a block
#define ORIENTATION_COMPUTE_PER_BLOCK 64
//how many keypoint descriptor to compute in a block (2, 4, 8, 16, 32)
#define DESCRIPTOR_COMPUTE_PER_BLOCK	4
#define DESCRIPTOR_COMPUTE_BLOCK_SIZE	(16 * DESCRIPTOR_COMPUTE_PER_BLOCK)
//how many keypoint descriptor to normalized in a block (32, ...)
#define DESCRIPTOR_NORMALIZ_PER_BLOCK	32



///////////////////////////////////////////
//Thread block size for visualization 
//(This doesn't affect the speed of computation)
#define BLOCK_LOG_DIM 4
#define BLOCK_DIM (1 << BLOCK_LOG_DIM)

//End SiftGPU setting section.
//----------------------------------------------------------------


__device__ __constant__ float d_kernel[KERNEL_MAX_WIDTH];
texture<float, 1, cudaReadModeElementType> texData;
texture<unsigned char, 1, cudaReadModeNormalizedFloat> texDataB;
texture<float2, 2, cudaReadModeElementType> texDataF2;
texture<float4, 1, cudaReadModeElementType> texDataF4;
texture<int4, 1, cudaReadModeElementType> texDataI4;
texture<int4, 1, cudaReadModeElementType> texDataList;

//template<int i>	 __device__ float Conv(float *data)		{    return Conv<i-1>(data) + data[i]*d_kernel[i];}
//template<>		__device__ float Conv<0>(float *data)	{    return data[0] * d_kernel[0];					}


//////////////////////////////////////////////////////////////
template<int FW> __global__ void FilterH( float* d_result, int width)
{

	const int HALF_WIDTH = FW >> 1; //    FW/2高斯卷积缩减一半
	const int CACHE_WIDTH = FILTERH_TILE_WIDTH + FW -1;  //128+FW-1     共享内存大小
	const int CACHE_COUNT = 2 + (CACHE_WIDTH - 2)/ FILTERH_TILE_WIDTH;   //一个catch包含几个高斯卷积核
	__shared__ float data[CACHE_WIDTH];  //128+一个高斯卷积核大小
	const int bcol = IMUL(blockIdx.x, FILTERH_TILE_WIDTH);  //128*blockIdx.x
	const int col =  bcol + threadIdx.x;  //线程索引
	const int index_min = IMUL(blockIdx.y, width);//每一行第一个
	const int index_max = index_min + width - 1;//每一行最后一个
	int src_index = index_min + bcol - HALF_WIDTH + threadIdx.x;  //每一行第blockIdx.x个线程块，减去高斯核的一半，有边缘效应！
	int cache_index = threadIdx.x; //0~128
	float value = 0;
#pragma unroll
	for(int j = 0; j < CACHE_COUNT; ++j)  //一个catch包含几个高斯卷积核
	{
		if(cache_index < CACHE_WIDTH)  //128+FW-128=FW  也就是前FW个线程运算了两次！！！
		{
			int fetch_index = src_index < index_min? index_min : (src_index > index_max ? index_max : src_index);
			data[cache_index] = tex1Dfetch(texData,fetch_index);
			src_index += FILTERH_TILE_WIDTH;
			cache_index += FILTERH_TILE_WIDTH;
		}
	}
	__syncthreads();  
	if(col >= width) return;
#pragma unroll
	for(int i = 0; i < FW; ++i)
	{
		value += (data[threadIdx.x + i]* d_kernel[i]);
	}
	//	value = Conv<FW-1>(data + threadIdx.x);
	d_result[index_min + col] = value;
}



////////////////////////////////////////////////////////////////////
template<int  FW>  __global__ void FilterV(float* d_result, int width, int height)
{
	const int HALF_WIDTH = FW >> 1;  //滤波的一半
	const int CACHE_WIDTH = FW + FILTERV_TILE_HEIGHT - 1;  //128+FW-1
	const int TEMP = CACHE_WIDTH & 0xf;//最大值是15
	//add some extra space to avoid bank conflict
#if FILTERV_TILE_WIDTH == 16
	//make the stride 16 * n +/- 1  步幅
	const int EXTRA = (TEMP == 1 || TEMP == 0) ? 1 - TEMP : 15 - TEMP;
#elif FILTERV_TILE_WIDTH == 8
	//make the stride 16 * n +/- 2
	const int EXTRA = (TEMP == 2 || TEMP == 1 || TEMP == 0) ? 2 - TEMP : (TEMP == 15? 3 : 14 - TEMP);
#elif FILTERV_TILE_WIDTH == 4
	//make the stride 16 * n +/- 4
	const int EXTRA = (TEMP >=0 && TEMP <=4) ? 4 - TEMP : (TEMP > 12? 20 - TEMP : 12 - TEMP);
#else
#error
#endif
	const int CACHE_TRUE_WIDTH = CACHE_WIDTH + EXTRA;//真正的共享内存宽度，
	const int CACHE_COUNT = (CACHE_WIDTH + FILTERV_BLOCK_HEIGHT - 1) / FILTERV_BLOCK_HEIGHT;  //catchlength/32
	const int WRITE_COUNT = (FILTERV_TILE_HEIGHT + FILTERV_BLOCK_HEIGHT -1) / FILTERV_BLOCK_HEIGHT;//128/32

	__shared__ float data[CACHE_TRUE_WIDTH * FILTERV_TILE_WIDTH];  //CACHE_TRUE_WIDTH*16
	const int row_block_first = IMUL(blockIdx.y, FILTERV_TILE_HEIGHT); //行索引的第一个值
	const int col = IMUL(blockIdx.x, FILTERV_TILE_WIDTH) + threadIdx.x; //这个才是正常的列索引
	const int row_first = row_block_first - HALF_WIDTH;//影像第一个行索引
	const int data_index_max = IMUL(height - 1, width) + col; //最后一行最后一个
	const int cache_col_start = threadIdx.y;	//列开始的地方
	const int cache_row_start = IMUL(threadIdx.x, CACHE_TRUE_WIDTH);
	int cache_index = cache_col_start + cache_row_start; //行列交叉
	int data_index = IMUL(row_first + cache_col_start, width) + col;//行列交叉

	if(col < width) 
	{
#pragma unroll
		for(int i = 0; i < CACHE_COUNT; ++i)
		{
			if(cache_col_start < CACHE_WIDTH - i * FILTERV_BLOCK_HEIGHT) 
			{
				int fetch_index = data_index < col ? col : (data_index > data_index_max? data_index_max : data_index);
				//如果把共享内存比作二位的，那么对应的就是左边一部分，在整个索引中分配是间断的！！！
				//随着循环的次数慢慢补齐你懂的
				data[cache_index + i * FILTERV_BLOCK_HEIGHT] = tex1Dfetch(texData,fetch_index);
				data_index += IMUL(FILTERV_BLOCK_HEIGHT, width);//每隔32行
			}
		}
	}
	__syncthreads();  //已完成共享内存的分配

	if(col >= width) return;

	int row = row_block_first + threadIdx.y;
	int index_start = cache_row_start + threadIdx.y;
#pragma unroll           //128/32                                             32                                                         32
	for(int i = 0; i < WRITE_COUNT;++i,row += FILTERV_BLOCK_HEIGHT, index_start += FILTERV_BLOCK_HEIGHT)
	{
		if(row < height)
		{
			int index_dest = IMUL(row, width) + col;
			float value = 0;
#pragma unroll
			for(int i = 0; i < FW; ++i)
			{
				value += (data[index_start + i] * d_kernel[i]);
			}
			d_result[index_dest] = value;
		}
	}
}


template<int LOG_SCALE> __global__ void UpsampleKernel(float* d_result, int width)
{
	const int SCALE = (1 << LOG_SCALE), SCALE_MASK = (SCALE - 1);
	const float INV_SCALE = 1.0f / (float(SCALE));
	int col = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;
	if(col >= width) return;

	int row = blockIdx.y >> LOG_SCALE; 
	int index = row * width + col;
	int dst_row = blockIdx.y;
	int dst_idx= (width * dst_row + col) * SCALE;
	int helper = blockIdx.y & SCALE_MASK; 
	if (helper)
	{
		float v11 = tex1Dfetch(texData, index);
		float v12 = tex1Dfetch(texData, index + 1);
		index += width;
		float v21 = tex1Dfetch(texData, index);
		float v22 = tex1Dfetch(texData, index + 1);
		float w1 = INV_SCALE * helper, w2 = 1.0 - w1;
		float v1 = (v21 * w1  + w2 * v11);
		float v2 = (v22 * w1  + w2 * v12);
		d_result[dst_idx] = v1;
#pragma unroll
		for(int i = 1; i < SCALE; ++i)
		{
			const float r2 = i * INV_SCALE;
			const float r1 = 1.0f - r2; 
			d_result[dst_idx +i] = v1 * r1 + v2 * r2;
		}
	}else
	{
		float v1 = tex1Dfetch(texData, index);
		float v2 = tex1Dfetch(texData, index + 1);
		d_result[dst_idx] = v1;
#pragma unroll
		for(int i = 1; i < SCALE; ++i)
		{
			const float r2 = i * INV_SCALE;
			const float r1 = 1.0f - r2; 
			d_result[dst_idx +i] = v1 * r1 + v2 * r2;
		}
	}

}

////////////////////////////////////////////////////////////////////////////////////////
void ProgramCU::SampleImageU(CuTexImage *dst, CuTexImage *src, int log_scale)
{
	int width = src->GetImgWidth(), height = src->GetImgHeight();
	src->BindTexture(texData);
	dim3 grid((width +  FILTERH_TILE_WIDTH - 1)/ FILTERH_TILE_WIDTH, height << log_scale);
	dim3 block(FILTERH_TILE_WIDTH);
	switch(log_scale)
	{
	case 1 : 	UpsampleKernel<1> <<< grid, block>>> ((float*) dst->_cuData, width);	break;
	case 2 : 	UpsampleKernel<2> <<< grid, block>>> ((float*) dst->_cuData, width);	break;
	case 3 : 	UpsampleKernel<3> <<< grid, block>>> ((float*) dst->_cuData, width);	break;
	default:	break;
	}
}

template<int LOG_SCALE> __global__ void DownsampleKernel(float* d_result, int src_width, int dst_width)
{
	const int dst_col = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;
	if(dst_col >= dst_width) return;
	const int src_col = min((dst_col << LOG_SCALE), (src_width - 1));  //dst_col*2
	const int dst_row = blockIdx.y;    //降采样影像
	const int src_row = blockIdx.y << LOG_SCALE;  //源影像  dst_row*2
	const int src_idx = IMUL(src_row, src_width) + src_col;
	const int dst_idx = IMUL(dst_width, dst_row) + dst_col;
	d_result[dst_idx] = tex1Dfetch(texData, src_idx);

}

__global__ void DownsampleKernel(float* d_result, int src_width, int dst_width, const int log_scale)
{
	const int dst_col = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;
	if(dst_col >= dst_width) return;
	const int src_col = min((dst_col << log_scale), (src_width - 1));
	const int dst_row = blockIdx.y; 
	const int src_row = blockIdx.y << log_scale;
	const int src_idx = IMUL(src_row, src_width) + src_col;
	const int dst_idx = IMUL(dst_width, dst_row) + dst_col;
	d_result[dst_idx] = tex1Dfetch(texData, src_idx);

}

void ProgramCU::SampleImageD(CuTexImage *dst, CuTexImage *src, int log_scale)
{
	int src_width = src->GetImgWidth(), dst_width = dst->GetImgWidth() ;

	src->BindTexture(texData);
	dim3 grid((dst_width +  FILTERH_TILE_WIDTH - 1)/ FILTERH_TILE_WIDTH, dst->GetImgHeight());
	dim3 block(FILTERH_TILE_WIDTH);
	switch(log_scale)
	{
	case 1 : 	DownsampleKernel<1> <<< grid, block>>> ((float*) dst->_cuData, src_width, dst_width);	break;
	case 2 :	DownsampleKernel<2> <<< grid, block>>> ((float*) dst->_cuData, src_width, dst_width);	break;
	case 3 : 	DownsampleKernel<3> <<< grid, block>>> ((float*) dst->_cuData, src_width, dst_width);	break;
	default:	DownsampleKernel    <<< grid, block>>> ((float*) dst->_cuData, src_width, dst_width, log_scale);
	}
		cudaThreadSynchronize();
}

__global__ void ChannelReduce_Kernel(float* d_result)
{
	int index = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;
	d_result[index] = tex1Dfetch(texData, index*4);
}

__global__ void ChannelReduce_Convert_Kernel(float* d_result)
{
	int index = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;
	float4 rgba = tex1Dfetch(texDataF4, index);
	d_result[index] = 0.299f * rgba.x + 0.587f* rgba.y + 0.114f * rgba.z;
}

void ProgramCU::ReduceToSingleChannel(CuTexImage* dst, CuTexImage* src, int convert_rgb)
{
	int width = src->GetImgWidth(), height = dst->GetImgHeight() ;

	dim3 grid((width * height +  FILTERH_TILE_WIDTH - 1)/ FILTERH_TILE_WIDTH);
	dim3 block(FILTERH_TILE_WIDTH);
	if(convert_rgb)
	{
		src->BindTexture(texDataF4);
		ChannelReduce_Convert_Kernel<<<grid, block>>>((float*)dst->_cuData);
	}else
	{
		src->BindTexture(texData);
		ChannelReduce_Kernel<<<grid, block>>>((float*)dst->_cuData);
	}
}

__global__ void ConvertByteToFloat_Kernel(float* d_result)
{
	int index = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;
	d_result[index] = tex1Dfetch(texDataB, index);
}

void ProgramCU::ConvertByteToFloat(CuTexImage*src, CuTexImage* dst)
{
	int width = src->GetImgWidth(), height = dst->GetImgHeight() ;
	dim3 grid((width * height +  FILTERH_TILE_WIDTH - 1)/ FILTERH_TILE_WIDTH);
	dim3 block(FILTERH_TILE_WIDTH);
	src->BindTexture(texDataB);
	ConvertByteToFloat_Kernel<<<grid, block>>>((float*)dst->_cuData);
}

void ProgramCU::CreateFilterKernel(float sigma, float* kernel, int& width)
{
	int i, sz = int( ceil( GlobalUtil::_FilterWidthFactor * sigma -0.5) ) ;//
	width = 2*sz + 1;

	if(width > KERNEL_MAX_WIDTH)
	{
		//filter size truncation
		sz = KERNEL_MAX_WIDTH >> 1;
		width =KERNEL_MAX_WIDTH;
	}else if(width < KERNEL_MIN_WIDTH)
	{
		sz = KERNEL_MIN_WIDTH >> 1;
		width =KERNEL_MIN_WIDTH;
	}

	float   rv = 1.0f/(sigma*sigma), v, ksum =0; 

	// pre-compute filter
	for( i = -sz ; i <= sz ; ++i) 
	{
		kernel[i+sz] =  v = exp(-0.5f * i * i *rv) ;
		ksum += v;
	}

	//normalize the kernel
	rv = 1.0f/ksum;
	for(i = 0; i< width ;i++) kernel[i]*=rv;
}


template<int FW> void ProgramCU::FilterImage(CuTexImage *dst, CuTexImage *src, CuTexImage* buf)
{
	int width = src->GetImgWidth(), height = src->GetImgHeight();






	//horizontal filtering
	src->BindTexture(texData); //src是源图像
	dim3 gridh((width +  FILTERH_TILE_WIDTH - 1)/ FILTERH_TILE_WIDTH, height);
	dim3 blockh(FILTERH_TILE_WIDTH);


			//GlobalUtil::StartTimer("水平");
	FilterH<FW><<<gridh, blockh>>>((float*)buf->_cuData, width);
	cudaThreadSynchronize();
	//	GlobalUtil::StopTimer();
	//	float _timing0 = GlobalUtil::GetElapsedTime();
	//CheckErrorCUDA("FilterH");




	///vertical filtering
	buf->BindTexture(texData);
	//16,128
	dim3 gridv((width + FILTERV_TILE_WIDTH - 1)/ FILTERV_TILE_WIDTH,  (height + FILTERV_TILE_HEIGHT - 1)/FILTERV_TILE_HEIGHT); //(50,5)
	dim3 blockv(FILTERV_TILE_WIDTH, FILTERV_BLOCK_HEIGHT);   //(16*32)
	//GlobalUtil::StartTimer("竖直");
	FilterV<FW><<<gridv, blockv>>>((float*)dst->_cuData, width, height); 
				cudaThreadSynchronize();			
		//		GlobalUtil::StopTimer();

		//float _timing1 = GlobalUtil::GetElapsedTime();
		
		
		//0.005,0.008   是原来的1.6倍！！！
	CheckErrorCUDA("FilterV");
}

//////////////////////////////////////////////////////////////////////
// tested on 2048x1500 image, the time on pyramid construction is
// OpenGL version : 18ms
// CUDA version: 28 ms
void ProgramCU::FilterImage(CuTexImage *dst, CuTexImage *src, CuTexImage* buf, float sigma)
{
	float filter_kernel[KERNEL_MAX_WIDTH]; int width;
	CreateFilterKernel(sigma, filter_kernel, width);
	cudaMemcpyToSymbol(d_kernel, filter_kernel, width * sizeof(float), 0, cudaMemcpyHostToDevice);

	switch(width)
	{
	case 5:		FilterImage< 5>(dst, src, buf);	break;
	case 7:		FilterImage< 7>(dst, src, buf);	break;
	case 9:		FilterImage< 9>(dst, src, buf);	break;
	case 11:	FilterImage<11>(dst, src, buf);	break;
	case 13:	FilterImage<13>(dst, src, buf);	break;
	case 15:	FilterImage<15>(dst, src, buf);	break;
	case 17:	FilterImage<17>(dst, src, buf);	break;
	case 19:	FilterImage<19>(dst, src, buf);	break;
	case 21:	FilterImage<21>(dst, src, buf);	break;
	case 23:	FilterImage<23>(dst, src, buf);	break;
	case 25:	FilterImage<25>(dst, src, buf);	break;
	case 27:	FilterImage<27>(dst, src, buf);	break;
	case 29:	FilterImage<29>(dst, src, buf);	break;
	case 31:	FilterImage<31>(dst, src, buf);	break;
	case 33:	FilterImage<33>(dst, src, buf);	break;
	default:	break;
	}

}


texture<float, 1, cudaReadModeElementType> texC;
texture<float, 1, cudaReadModeElementType> texP;
texture<float, 1, cudaReadModeElementType> texN;

void __global__ ComputeDOG_Kernel(float* d_dog, float2* d_got, int width, int height)
{
	int row = (blockIdx.y << DOG_BLOCK_LOG_DIMY) + threadIdx.y;
	int col = (blockIdx.x << DOG_BLOCK_LOG_DIMX) + threadIdx.x;
	if(col < width && row < height) 
	{
		int index = IMUL(row, width) + col;
		float vp = tex1Dfetch(texP, index);
		float v = tex1Dfetch(texC, index);
		d_dog[index] = v - vp;
		float vxn = tex1Dfetch(texC, index + 1);
		float vxp = tex1Dfetch(texC, index - 1);
		float vyp = tex1Dfetch(texC, index - width);
		float vyn = tex1Dfetch(texC, index + width);
		float dx = vxn - vxp, dy = vyn - vyp;
		float grd = 0.5f * sqrt(dx * dx  + dy * dy);
		float rot = (grd == 0.0f? 0.0f : atan2(dy, dx));
		d_got[index] = make_float2(grd, rot);
	}
}

void __global__ ComputeDOG_Kernel(float* d_dog, int width, int height)
{
	int row = (blockIdx.y << DOG_BLOCK_LOG_DIMY) + threadIdx.y;
	int col = (blockIdx.x << DOG_BLOCK_LOG_DIMX) + threadIdx.x;
	if(col < width && row < height) 
	{
		int index = IMUL(row, width) + col;
		float vp = tex1Dfetch(texP, index);
		float v = tex1Dfetch(texC, index);
		d_dog[index] = v - vp;
	}
}

void ProgramCU::ComputeDOG(CuTexImage* gus, CuTexImage* dog, CuTexImage* got)
{
	int width = gus->GetImgWidth(), height = gus->GetImgHeight();
	dim3 grid((width + DOG_BLOCK_DIMX - 1)/ DOG_BLOCK_DIMX,  (height + DOG_BLOCK_DIMY - 1)/DOG_BLOCK_DIMY);
	dim3 block(DOG_BLOCK_DIMX, DOG_BLOCK_DIMY);
	gus->BindTexture(texC);
	(gus -1)->BindTexture(texP);  //got实际上是前三层高斯金字塔的梯度值
	if(got->_cuData)
		ComputeDOG_Kernel<<<grid, block>>>((float*) dog->_cuData, (float2*) got->_cuData, width, height);
	else
		ComputeDOG_Kernel<<<grid, block>>>((float*) dog->_cuData, width, height);
	cudaThreadSynchronize();
}


#define READ_CMP_DOG_DATA(datai, tex, idx) \
	datai[0] = tex1Dfetch(tex, idx - 1);\
	datai[1] = tex1Dfetch(tex, idx);\
	datai[2] = tex1Dfetch(tex, idx + 1);\
	if(v > nmax)\
{\
	nmax = max(nmax, datai[0]);\
	nmax = max(nmax, datai[1]);\
	nmax = max(nmax, datai[2]);\
	if(v < nmax) goto key_finish;\
}else\
{\
	nmin = min(nmin, datai[0]);\
	nmin = min(nmin, datai[1]);\
	nmin = min(nmin, datai[2]);\
	if(v > nmin) goto key_finish;\
}


void __global__ ComputeKEY_Kernel(float4* d_key, int width, int colmax, int rowmax, 
	float dog_threshold0,  float dog_threshold, float edge_threshold, int subpixel_localization)
{
	float data[3][3], v;
	float datap[3][3], datan[3][3];
#ifdef KEY_OFFSET_ONE
	int row = (blockIdx.y << KEY_BLOCK_LOG_DIMY) + threadIdx.y + 1;
	int col = (blockIdx.x << KEY_BLOCK_LOG_DIMX) + threadIdx.x + 1;
#else
	int row = (blockIdx.y << KEY_BLOCK_LOG_DIMY) + threadIdx.y;
	int col = (blockIdx.x << KEY_BLOCK_LOG_DIMX) + threadIdx.x;
#endif
	int index = IMUL(row, width) + col;
	int idx[3] ={index - width, index, index + width};
	int in_image =0;
	float nmax, nmin, result = 0.0f;
	float dx = 0, dy = 0, ds = 0;
	bool offset_test_passed = true;
#ifdef KEY_OFFSET_ONE
	if(row < rowmax && col < colmax)
#else
	if(row > 0 && col > 0 && row < rowmax && col < colmax)
#endif
	{
		//一维抑制！！！！！！！
		in_image = 1;
		data[1][1] = v = tex1Dfetch(texC, idx[1]);  //当前像元
		if(fabs(v) <= dog_threshold0) goto key_finish;

		data[1][0] = tex1Dfetch(texC, idx[1] - 1);
		data[1][2] = tex1Dfetch(texC, idx[1] + 1);//左右像元
		nmax = max(data[1][0], data[1][2]);
		nmin = min(data[1][0], data[1][2]);//左右像元最大最小值

		if(v <=nmax && v >= nmin) goto key_finish; //极值是什么，是比所有值都大！！！当然比旁边的大，比旁边小的走开。
		//if((v > nmax && v < 0 )|| (v < nmin && v > 0)) goto key_finish;
		READ_CMP_DOG_DATA(data[0], texC, idx[0]);//第一行最大最小值
		READ_CMP_DOG_DATA(data[2], texC, idx[2]);//第三行最大最小值

		//二维抑制！！！！
		//edge supression  边缘检测！
		float vx2 = v * 2.0f;
		float fxx = data[1][0] + data[1][2] - vx2;
		float fyy = data[0][1] + data[2][1] - vx2;
		float fxy = 0.25f * (data[2][2] + data[0][0] - data[2][0] - data[0][2]);
		float temp1 = fxx * fyy - fxy * fxy;
		float temp2 = (fxx + fyy) * (fxx + fyy);
		if(temp1 <=0 || temp2 > edge_threshold * temp1) goto key_finish;
		
		//三维抑制！！！！
		//read the previous level
		READ_CMP_DOG_DATA(datap[0], texP, idx[0]);//上一层的最大最小值
		READ_CMP_DOG_DATA(datap[1], texP, idx[1]);
		READ_CMP_DOG_DATA(datap[2], texP, idx[2]);


		//read the next level
		READ_CMP_DOG_DATA(datan[0], texN, idx[0]);//下一层的最大最小值
		READ_CMP_DOG_DATA(datan[1], texN, idx[1]);
		READ_CMP_DOG_DATA(datan[2], texN, idx[2]);

		if(subpixel_localization)
		{
			//subpixel localization
			float fx = 0.5f * (data[1][2] - data[1][0]);
			float fy = 0.5f * (data[2][1] - data[0][1]);
			float fs = 0.5f * (datan[1][1] - datap[1][1]);

			float fss = (datan[1][1] + datap[1][1] - vx2);
			float fxs = 0.25f* (datan[1][2] + datap[1][0] - datan[1][0] - datap[1][2]);
			float fys = 0.25f* (datan[2][1] + datap[0][1] - datan[0][1] - datap[2][1]);

			//need to solve dx, dy, ds;
			// |-fx|     | fxx fxy fxs |   |dx|
			// |-fy|  =  | fxy fyy fys | * |dy|
			// |-fs|     | fxs fys fss |   |ds|
			float4 A0 = fxx > 0? make_float4(fxx, fxy, fxs, -fx) : make_float4(-fxx, -fxy, -fxs, fx);
			float4 A1 = fxy > 0? make_float4(fxy, fyy, fys, -fy) : make_float4(-fxy, -fyy, -fys, fy);
			float4 A2 = fxs > 0? make_float4(fxs, fys, fss, -fs) : make_float4(-fxs, -fys, -fss, fs);
			//高斯消元法
			float maxa = max(max(A0.x, A1.x), A2.x);  //选主元
			if(maxa >= 1e-10)
			{
				if(maxa == A1.x)//如果A1是最大的，那么A1和A0调换
				{
					float4 TEMP = A1; A1 = A0; A0 = TEMP;  
				}else if(maxa == A2.x)//如果A2是最大的，那么A1和A0调换
				{
					float4 TEMP = A2; A2 = A0; A0 = TEMP;
				}
				A0.y /= A0.x;	A0.z /= A0.x;	A0.w/= A0.x;
				A1.y -= A1.x * A0.y;	A1.z -= A1.x * A0.z;	A1.w -= A1.x * A0.w;
				A2.y -= A2.x * A0.y;	A2.z -= A2.x * A0.z;	A2.w -= A2.x * A0.w;
				if(abs(A2.y) > abs(A1.y))
				{
					float4 TEMP = A2;	A2 = A1; A1 = TEMP;
				}
				if(abs(A1.y) >= 1e-10) 
				{
					A1.z /= A1.y;	A1.w /= A1.y;
					A2.z -= A2.y * A1.z;	A2.w -= A2.y * A1.w;
					if(abs(A2.z) >= 1e-10) 
					{
						ds = A2.w / A2.z;
						dy = A1.w - ds * A1.z;
						dx = A0.w - ds * A0.z - dy * A0.y;

						offset_test_passed = 
							fabs(data[1][1] + 0.5f * (dx * fx + dy * fy + ds * fs)) > dog_threshold   //去除低对比度的点
							&&fabs(ds) < 1.0f && fabs(dx) < 1.0f && fabs(dy) < 1.0f;
					}
				}
			}
		}
		if(offset_test_passed) result = v > nmax ? 1.0 : -1.0;//当前像元通过测试，大于最大值为极大值，否则为极小值！
	}
key_finish:  //已经知道位置了index就代表了行列，但是保存不了那么多信息
		if(in_image) d_key[index] = make_float4(result, dx, dy, ds);  //得到像元的改正值！(用周围像元替代该像元直到收敛，应该是以后的事）
}


void ProgramCU::ComputeKEY(CuTexImage* dog, CuTexImage* key, float Tdog, float Tedge)
{
	int width = dog->GetImgWidth(), height = dog->GetImgHeight();
	float Tdog1 = (GlobalUtil::_SubpixelLocalization? 0.8f : 1.0f) * Tdog;
	CuTexImage* dogp = dog - 1;
	CuTexImage* dogn = dog + 1;
#ifdef KEY_OFFSET_ONE
	dim3 grid((width - 1 + KEY_BLOCK_DIMX - 1)/ KEY_BLOCK_DIMX,  (height - 1 + KEY_BLOCK_DIMY - 1)/KEY_BLOCK_DIMY);
#else
	dim3 grid((width + KEY_BLOCK_DIMX - 1)/ KEY_BLOCK_DIMX,  (height + KEY_BLOCK_DIMY - 1)/KEY_BLOCK_DIMY);
#endif
	dim3 block(KEY_BLOCK_DIMX, KEY_BLOCK_DIMY);
	dogp->BindTexture(texP);
	dog ->BindTexture(texC);
	dogn->BindTexture(texN);
	Tedge = (Tedge+1)*(Tedge+1)/Tedge;
	//(8,8) (800/8,600/8)
	ComputeKEY_Kernel<<<grid, block>>>((float4*) key->_cuData, width,
		width -1, height -1, Tdog1, Tdog, Tedge, GlobalUtil::_SubpixelLocalization);
		cudaThreadSynchronize();
}


//ws 800,wd 200,height 600
void __global__ InitHist_Kernel(int4* hist, int ws, int wd, int height)
{
	int row = IMUL(blockIdx.y, blockDim.y) + threadIdx.y;//行索引
	int col = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;//列索引
	if(row < height && col < wd)
	{
		int hidx = IMUL(row, wd) + col; //直方图索引(0,1,2,3,4...200)
		int scol = col << 2;//乘以4  (0,4,8,12...800)
		int sidx = IMUL(row, ws) + scol;//影像索引(0,4,8,12...800)
		int v[4] = {0, 0, 0, 0}; 
		if(row > 0 && row < height -1)
		{
#pragma unroll
			for(int i = 0; i < 4 ; ++i, ++scol)
			{
				float4 temp = tex1Dfetch(texDataF4, sidx +i);//当前像元以及右边三个
				//temp(result, dx, dy, ds)，result不为0说明是特征点！
				v[i] = (scol < ws -1 && scol > 0 && temp.x!=0) ? 1 : 0;//满足条件，不超过列索引，且temp.x不为零       则为1
			}
		}
		hist[hidx] = make_int4(v[0], v[1], v[2], v[3]);//高度不变，宽度变为之前的1/4

	}
}



void ProgramCU::InitHistogram(CuTexImage* key, CuTexImage* hist)
{
	int ws = key->GetImgWidth(), hs = key->GetImgHeight();//800*600
	int wd = hist->GetImgWidth(), hd = hist->GetImgHeight();//200*600
	dim3 grid((wd  + HIST_INIT_WIDTH - 1)/ HIST_INIT_WIDTH,  hd);//(200/128,600)
	dim3 block(HIST_INIT_WIDTH, 1);  //(128,1)
	key->BindTexture(texDataF4);
	//hist->cuda,800,200,600
	InitHist_Kernel<<<grid, block>>>((int4*) hist->_cuData, ws, wd, hd);
		cudaThreadSynchronize();
}


//200,50,600
void __global__ ReduceHist_Kernel(int4* d_hist, int ws, int wd, int height)
{
	int row = IMUL(blockIdx.y, blockDim.y) + threadIdx.y;//行索引
	int col = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;//列索引
	if(row < height && col < wd)
	{
		int hidx = IMUL(row, wd) + col;//直方图索引
		int scol = col << 2;//col*4
		int sidx = IMUL(row, ws) + scol;//上一层直方图索引
		int v[4] = {0, 0, 0, 0}; 
#pragma unroll
		for(int i = 0; i < 4 && scol < ws; ++i, ++scol)
		{
			int4 temp = tex1Dfetch(texDataI4, sidx + i);//上一层直方图中的像元和右边三个像元  200--50四倍关系！
			v[i] = temp.x + temp.y + temp.z + temp.w; //直方图的四个元素(x,y,z,w)
		}
		d_hist[hidx] = make_int4(v[0], v[1], v[2], v[3]);//本层的直方图索引
	}
}

void ProgramCU::ReduceHistogram(CuTexImage*hist1, CuTexImage* hist2)
{
	int ws = hist1->GetImgWidth(), hs = hist1->GetImgHeight();
	int wd = hist2->GetImgWidth(), hd = hist2->GetImgHeight();
	int temp = (int)floor(logf(float(wd * 2/ 3)) / logf(2.0f));
	const int wi = min(7, max(temp , 0));
	hist1->BindTexture(texDataI4);

	const int BW = 1 << wi, BH =  1 << (7 - wi);
	dim3 grid((wd  + BW - 1)/ BW,  (hd + BH -1) / BH);//（wd/32,hd/4）
	dim3 block(BW, BH);//（32,4）
	ReduceHist_Kernel<<<grid, block>>>((int4*)hist2->_cuData, ws, wd, hd);
		cudaThreadSynchronize();
}


void __global__ ListGen_Kernel(int4* d_list, int width)
{
	int idx1 = IMUL(blockIdx.x, blockDim.x) + threadIdx.x; //特征点索引
	int4 pos = tex1Dfetch(texDataList, idx1);  //拾取纹理内存
	int idx2 = IMUL(pos.y, width) + pos.x; //直方图索引（不为0的）   y指600行中的第几行，即第几个特征点，x为特征点具体位置，第一次width为4
	int4 temp = tex1Dfetch(texDataI4, idx2);//拾取直方图纹理内存
	int  sum1 = temp.x + temp.y;
	int  sum2 = sum1 + temp.z;                                                                                                                                                             
	pos.x <<= 2;//pos.x *4
	if(pos.z >= sum2)//这个设计的好巧妙呀！！！  先算pos.x=0的，再把pos.x不等于0的化为0，再按照pos.x=0的情况算
	{
		pos.x += 3;
		pos.z -= sum2;
	}else if(pos.z >= sum1)
	{
		pos.x += 2;
		pos.z -= sum1;
	}else if(pos.z >= temp.x)
	{
		pos.x += 1;
		pos.z -= temp.x;
	}
	d_list[idx1] = pos;
}

//input list (x, y) (x, y) ....           特征feature层，hist层直方图层
void ProgramCU::GenerateList(CuTexImage* list, CuTexImage* hist)
{
	int len = list->GetImgWidth();//327个特征点
	list->BindTexture(texDataList); 
	hist->BindTexture(texDataI4);
	dim3  grid((len + LISTGEN_BLOCK_DIM -1) /LISTGEN_BLOCK_DIM);//  len/128
	dim3  block(LISTGEN_BLOCK_DIM);//128
	//listgenerate，列表生成核函数
	ListGen_Kernel<<<grid, block>>>((int4*) list->_cuData, hist->GetImgWidth());
		cudaThreadSynchronize();
}

void __global__ ComputeOrientation_Kernel(float4* d_list, 
	int list_len,//327
	int width, int height,  //800,600
	float sigma, float sigma_step, //2.01,1.26
	float gaussian_factor, float sample_factor,//1.5,3
	int num_orientation,//2
	int existing_keypoint, //0
	int subpixel, //1
	int keepsign)//0
{
	//10度每半径
	const float ten_degree_per_radius = 5.7295779513082320876798154814105; //(360/10)/2pi
	//半径每十度
	const float radius_per_ten_degrees = 1.0 / 5.7295779513082320876798154814105;//(10/360)*2PI
	int idx = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;//线程索引
	if(idx >= list_len) return;
	float4 key; //重构key！！！(x,y,sigma,)
	if(existing_keypoint)
	{
		key = tex1Dfetch(texDataF4, idx);		
	}else 
	{
		int4 ikey = tex1Dfetch(texDataList, idx);//(x,y,sigma,)
		key.x = ikey.x + 0.5f; //四舍五入
		key.y = ikey.y + 0.5f;
		key.z = sigma;
		if(subpixel || keepsign)
		{
			float4 offset = tex1Dfetch(texDataF4, IMUL(width, ikey.y) + ikey.x);//定位到关键点
			if(subpixel)
			{
				//key（result,dx,dy,ds）
				key.x += offset.y;//x+=dx
				key.y += offset.z;//y+=dy
				key.z *= pow(sigma_step, offset.w);//z*=dz
			}
			if(keepsign) key.z *= offset.x; //???
		}
	}
	if(num_orientation == 0)
	{
		key.w = 0;
		d_list[idx] = key;
		return;
	}
	float vote[37]; //权！！！
	float gsigma = key.z * gaussian_factor;//key.z就是sigma
	float win = fabs(key.z) * sample_factor;//窗口
	float dist_threshold = win * win + 0.5;  //距离阈值
	float factor = -0.5f / (gsigma * gsigma);
	float xmin = max(1.5f, floor(key.x - win) + 0.5f);  //-radius
	float ymin = max(1.5f, floor(key.y - win) + 0.5f);
	float xmax = min(width - 1.5f, floor(key.x + win) + 0.5f);//+radius
	float ymax = min(height -1.5f, floor(key.y + win) + 0.5f);
#pragma unroll
	for(int i = 0; i < 36; ++i) vote[i] = 0.0f;  //36个方向
	for(float y = ymin; y <= ymax; y += 1.0f) //-radius~+radius
	{
		for(float x = xmin; x <= xmax; x += 1.0f)//-radius~+radius
		{
			float dx = x - key.x;  //i
			float dy = y - key.y; //j
			float sq_dist  = dx * dx + dy * dy;

			float2 got = tex2D(texDataF2, x, y);// 二维纹理  got.x是梯度幅值，got.y是梯度方向
			float weight = got.x * exp(sq_dist * factor);//该点相对于特征点的高斯权重！！！   这个是梯度的幅值
			float fidx = floor(got.y * ten_degree_per_radius);//不超过2PI的乘以5肯定不超过36！
			int oidx = fidx;
			if(oidx < 0) oidx += 36;
			vote[oidx] += weight; //可能有很多个的！！！累加到这个方向
		}
	}

	//filter the vote   高斯权滤波

	const float one_third = 1.0 /3.0;//三分之一
#pragma unroll  //循环展开
	for(int i = 0; i < 6; ++i)
	{
		vote[36] = vote[0];
		float pre = vote[35];
#pragma unroll
		for(int j = 0; j < 36; ++j)
		{
			float temp = one_third * (pre + vote[j] + vote[j + 1]);
			pre = vote[j];			vote[j] = temp;
			//1、当前值赋值给pre2、当前值为左边值和右边值中间值平均值
		}
	}

	vote[36] = vote[0];
	if(num_orientation == 1 || existing_keypoint)
	{
		int index_max = 0;
		float max_vote = vote[0];
#pragma unroll
		for(int i = 1; i < 36; ++i)
		{
			index_max =  vote[i] > max_vote? i : index_max;
			max_vote = max(max_vote, vote[i]);
		}
		float pre = vote[index_max == 0? 35 : index_max -1];
		float next = vote[index_max + 1];
		float weight = max_vote;
		float off =  0.5f * FDIV(next - pre, weight + weight - next - pre);
		key.w = radius_per_ten_degrees * (index_max + 0.5f + off);
		d_list[idx] = key;

	}else
	{
		float max_vote = vote[0];
#pragma unroll
		for(int i = 1; i < 36; ++i)		max_vote = max(max_vote, vote[i]);  //找到最大权值

		float vote_threshold = max_vote * 0.8f;  //80% 峰值！！
		float pre = vote[35];
		float max_rot[2], max_vot[2] = {0, 0};  //主方向和辅方向
		int  ocount = 0;
#pragma unroll
		for(int i =0; i < 36; ++i)//36个方向
		{
			float next = vote[i + 1];//下一个
			if(vote[i] > vote_threshold && vote[i] > pre && vote[i] > next)
			{
				float di = 0.5f * FDIV(next - pre, vote[i] + vote[i] - next - pre);//除以
				float rot = i + di + 0.5f;
				float weight = vote[i];
				///得到的结果是max_vot[0]是最大的    max_vot[1]>=max_vot[0]
				if(weight > max_vot[1])
				{
					if(weight > max_vot[0])
					{
						max_vot[1] = max_vot[0]; //
						max_rot[1] = max_rot[0];
						max_vot[0] = weight;
						max_rot[0] = rot;
					}
					else
					{
						max_vot[1] = weight;
						max_rot[1] = rot;
					}
					ocount ++;
				}
			}
			pre = vote[i];
		}
		float fr1 = max_rot[0] / 36.0f; //归一化（0,1）主方向  得出的是一个百分比！
		if(fr1 < 0) fr1 += 1.0f; 
		unsigned short us1 = ocount == 0? 65535 : ((unsigned short )floor(fr1 * 65535.0f));
		unsigned short us2 = 65535; 
		if(ocount > 1)  //如果ocount为1，则说明没有辅助方向，大于1则说明有辅方向
		{
			float fr2 = max_rot[1] / 36.0f; //归一化（0,1）
			if(fr2 < 0) fr2 += 1.0f;
			us2 = (unsigned short ) floor(fr2 * 65535.0f);
		}
		unsigned int uspack = (us2 << 16) | us1; //us2*2^16   us2移位运算!!! 保留了两个方向！！！
		//数学不好是坑啊，3二进制是11,3*2=6二进制是110，相当于每乘以2后面加一个0
		//也就是加了16个0，也就是避开了65535！保留了主方向和辅助方向！
		key.w = __int_as_float(uspack);//把int作为float保存在key.w中，别忘了key的数据类型是！！！float4
		d_list[idx] = key;
	}

}




void ProgramCU::ComputeOrientation(CuTexImage* list, CuTexImage* got, CuTexImage*key, 
	float sigma, float sigma_step, int existing_keypoint)
{
	int len = list->GetImgWidth();
	if(len <= 0) return;
	int width = got->GetImgWidth(), height = got->GetImgHeight();
	if(existing_keypoint)
	{
		list->BindTexture(texDataF4);
	}else
	{
		list->BindTexture(texDataList);
		if(GlobalUtil::_SubpixelLocalization) key->BindTexture(texDataF4);//F4(result,dx,dy,ds)
	}
	got->BindTexture2D(texDataF2);  //F2,梯度和角度

	const int block_width = len < ORIENTATION_COMPUTE_PER_BLOCK ? 16 : ORIENTATION_COMPUTE_PER_BLOCK;   //len<64,以16位大小，大于以64
	dim3 grid((len + block_width -1) / block_width);
	dim3 block(block_width);

	ComputeOrientation_Kernel<<<grid, block>>>((float4*) list->_cuData, 
		len, width, height, sigma, sigma_step, 
		GlobalUtil::_OrientationGaussianFactor, 
		GlobalUtil::_OrientationGaussianFactor * GlobalUtil::_OrientationWindowFactor,
		GlobalUtil::_FixedOrientation? 0 : GlobalUtil::_MaxOrientation, //0是假
		existing_keypoint, GlobalUtil::_SubpixelLocalization, GlobalUtil::_KeepExtremumSign);
		cudaThreadSynchronize();

	ProgramCU::CheckErrorCUDA("ComputeOrientation");
}

template <bool DYNAMIC_INDEXING> void __global__ ComputeDescriptor_Kernel(float4* d_des, int num, 
	int width, int height, float window_factor)
{
	const float rpi = 4.0/ 3.14159265358979323846;
	int idx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	int fidx = idx >> 4;  //idx/16 得到真正的特征点个数
	if(fidx >= num) return;
	float4 key = tex1Dfetch(texDataF4, fidx);//featureTex特征点定位
	int bidx = idx& 0xf, ix = bidx & 0x3, iy = bidx >> 2;
	//(0,0),(0,1)(0,2)...(3,3)
	float spt = fabs(key.z * window_factor);//窗口大小  3sigma
	float s, c; __sincosf(key.w, &s, &c);  //旋转到主方向！！！
	float anglef = key.w > 3.14159265358979323846? key.w - (2.0 * 3.14159265358979323846) : key.w ; 
	float cspt = c * spt, sspt = s * spt;//单独看并没有什么意义
	float crspt = c / spt, srspt = s / spt;//单独看并没有什么意义
	float2 offsetpt, pt;
	float xmin, ymin, xmax, ymax, bsz;
	offsetpt.x = ix - 1.5f;//哦~~这个1.5好熟悉呀，原来是那个+d/2-0.5
	offsetpt.y = iy - 1.5f;//将(0~3)->转变为(-1.5~1.5)
	//种子点的坐标
	pt.x = cspt * offsetpt.x - sspt * offsetpt.y + key.x;  //关键点旋转后的坐标？
	pt.y = cspt * offsetpt.y + sspt * offsetpt.x + key.y;
	bsz =  fabs(cspt) + fabs(sspt); //radius  √2*spt
	xmin = max(1.5f, floor(pt.x - bsz) + 0.5f); //-radius~radius
	ymin = max(1.5f, floor(pt.y - bsz) + 0.5f); //-radius~radius
	xmax = min(width - 1.5f, floor(pt.x + bsz) + 0.5f);
	ymax = min(height - 1.5f, floor(pt.y + bsz) + 0.5f);
	float des[9];
#pragma unroll
	for(int i =0; i < 9; ++i) des[i] = 0.0f;
	//前面不管，已经确定了邻域窗口大小了
	for(float y = ymin; y <= ymax; y += 1.0f)
	{
		for(float x = xmin; x <= xmax; x += 1.0f)
		{
			float dx = x - pt.x;   //pt为中心点坐标
			float dy = y - pt.y;
			float nx = crspt * dx + srspt * dy; //得到区域坐标
			float ny = crspt * dy - srspt * dx;
			float nxn = fabs(nx);
			float nyn = fabs(ny);
			if(nxn < 1.0f && nyn < 1.0f)
			{
				float2 cc = tex2D(texDataF2, x, y);  //梯度金字塔
				float dnx = nx + offsetpt.x;
				float dny = ny + offsetpt.y;//又旋转回来了！！！
				float ww = exp(-0.125f * (dnx * dnx + dny * dny));
				float wx = 1.0 - nxn;
				float wy = 1.0 - nyn;
				float weight = ww * wx * wy * cc.x;
				float theta = (anglef - cc.y) * rpi; //旋转到主方向
				if(theta < 0) theta += 8.0f;
				float fo = floor(theta);
				int fidx = fo;//方向
				float weight1 = fo + 1.0f  - theta;  //1-(theta - fo)
				float weight2 = theta - fo;   //theta - fo
				if(DYNAMIC_INDEXING)
				{
					des[fidx] += (weight1 * weight);
					des[fidx + 1] += (weight2 * weight);
					//this dynamic indexing part might be slow
				}else
				{
#pragma unroll
					for(int k = 0; k < 8; ++k)
					{
						if(k == fidx) 
						{
							des[k] += (weight1 * weight);
							des[k+1] += (weight2 * weight);
						}
					}
				}
			}
		}
	}
	des[0] += des[8];

	int didx = idx << 1;//0,2,4,6,8
	//每个特征点八个方向，16个特征区域，16*8=128
	//32个idx为一个特征点！！！！
	//128(float)*num->32(float4)*num->2(float4)*  16*num



	//16个线程，16<<1=16*2  每个线程计算8个方向
	d_des[didx] = make_float4(des[0], des[1], des[2], des[3]);
	d_des[didx+1] = make_float4(des[4], des[5], des[6], des[7]);
}


template <bool DYNAMIC_INDEXING> void __global__ ComputeDescriptorRECT_Kernel(float4* d_des, int num, 
	int width, int height, float window_factor)
{
	const float rpi = 4.0/ 3.14159265358979323846;
	int idx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	int fidx = idx >> 4;
	if(fidx >= num) return;
	float4 key = tex1Dfetch(texDataF4, fidx);
	int bidx = idx& 0xf, ix = bidx & 0x3, iy = bidx >> 2;
	//float aspect_ratio = key.w / key.z;
	//float aspect_sq = aspect_ratio * aspect_ratio;
	float sptx = key.z * 0.25, spty = key.w * 0.25;
	float xmin, ymin, xmax, ymax; float2 pt;
	pt.x = sptx * (ix + 0.5f)  + key.x;
	pt.y = spty * (iy + 0.5f)  + key.y;
	xmin = max(1.5f, floor(pt.x - sptx) + 0.5f);
	ymin = max(1.5f, floor(pt.y - spty) + 0.5f);
	xmax = min(width - 1.5f, floor(pt.x + sptx) + 0.5f);
	ymax = min(height - 1.5f, floor(pt.y + spty) + 0.5f);
	float des[9];
#pragma unroll
	for(int i =0; i < 9; ++i) des[i] = 0.0f;
	for(float y = ymin; y <= ymax; y += 1.0f)
	{
		for(float x = xmin; x <= xmax; x += 1.0f)
		{
			float nx = (x - pt.x) / sptx;
			float ny = (y - pt.y) / spty;
			float nxn = fabs(nx);
			float nyn = fabs(ny);
			if(nxn < 1.0f && nyn < 1.0f)
			{
				float2 cc = tex2D(texDataF2, x, y);
				float wx = 1.0 - nxn;
				float wy = 1.0 - nyn;
				float weight =  wx * wy * cc.x;
				float theta = (- cc.y) * rpi;
				if(theta < 0) theta += 8.0f;
				float fo = floor(theta);
				int fidx = fo;
				float weight1 = fo + 1.0f  - theta;
				float weight2 = theta - fo;
				if(DYNAMIC_INDEXING)
				{
					des[fidx] += (weight1 * weight);
					des[fidx + 1] += (weight2 * weight);
					//this dynamic indexing part might be slow
				}else
				{
#pragma unroll
					for(int k = 0; k < 8; ++k)
					{
						if(k == fidx) 
						{
							des[k] += (weight1 * weight);
							des[k+1] += (weight2 * weight);
						}
					}
				}
			}
		}
	}
	des[0] += des[8];

	int didx = idx << 1;
	d_des[didx] = make_float4(des[0], des[1], des[2], des[3]);
	d_des[didx+1] = make_float4(des[4], des[5], des[6], des[7]);
}

void __global__ NormalizeDescriptor_Kernel(float4* d_des, int num)
{
	float4 temp[32];
	int idx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	if(idx >= num) return;
	int sidx = idx << 5;//idx*32
	float norm1 = 0, norm2 = 0;
#pragma unroll
	for(int i = 0; i < 32; ++i)  //32*4=128
	{
		temp[i] = tex1Dfetch(texDataF4, sidx +i);
		norm1 += (temp[i].x * temp[i].x + temp[i].y * temp[i].y +
			temp[i].z * temp[i].z + temp[i].w * temp[i].w);
	}
	norm1 = rsqrt(norm1);  //分母

#pragma unroll
	for(int i = 0; i < 32; ++i)
	{
		temp[i].x = min(0.2f, temp[i].x * norm1);
		temp[i].y = min(0.2f, temp[i].y * norm1);
		temp[i].z = min(0.2f, temp[i].z * norm1);
		temp[i].w = min(0.2f, temp[i].w * norm1);
		norm2 += (temp[i].x * temp[i].x + temp[i].y * temp[i].y +
			temp[i].z * temp[i].z + temp[i].w * temp[i].w);//
	}

	norm2 = rsqrt(norm2);
#pragma unroll
	for(int i = 0; i < 32; ++i)
	{
		temp[i].x *= norm2;		temp[i].y *= norm2;
		temp[i].z *= norm2;		temp[i].w *= norm2;
		d_des[sidx + i] = temp[i];
	}
}

void ProgramCU::ComputeDescriptor(CuTexImage*list, CuTexImage* got, CuTexImage* dtex, int rect, int stream)
{
	int num = list->GetImgWidth();
	int width = got->GetImgWidth();
	int height = got->GetImgHeight();

	dtex->InitTexture(num * 128, 1, 1);
	got->BindTexture2D(texDataF2);
	list->BindTexture(texDataF4);
	int block_width = DESCRIPTOR_COMPUTE_BLOCK_SIZE;//64
	dim3 grid((num * 16 + block_width -1) / block_width);//num*16/64    16*8=128
	dim3 block(block_width);//64

	if(rect)
	{
		if(GlobalUtil::_UseDynamicIndexing)
			ComputeDescriptorRECT_Kernel<true><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
		else
			ComputeDescriptorRECT_Kernel<false><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);

	}else
	{
		if(GlobalUtil::_UseDynamicIndexing)
			ComputeDescriptor_Kernel<true><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
		else
			ComputeDescriptor_Kernel<false><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
	}
			cudaThreadSynchronize();
	if(GlobalUtil::_NormalizedSIFT)
	{
		dtex->BindTexture(texDataF4);
		const int block_width = DESCRIPTOR_NORMALIZ_PER_BLOCK;
		dim3 grid((num + block_width -1) / block_width);
		dim3 block(block_width);
		NormalizeDescriptor_Kernel<<<grid, block>>>((float4*) dtex->_cuData, num);
				cudaThreadSynchronize();
	}
	CheckErrorCUDA("ComputeDescriptor");
}

//////////////////////////////////////////////////////
void ProgramCU::FinishCUDA()
{
	cudaThreadSynchronize();
}

int ProgramCU::CheckErrorCUDA(const char* location)
{
	cudaError_t e = cudaGetLastError();
	if(e)
	{
		if(location) fprintf(stderr, "%s:\t",  location);
		fprintf(stderr, "%s\n",  cudaGetErrorString(e));
		//assert(0);
		return 1;
	}else
	{
		return 0; 
	}
}

void __global__ ConvertDOG_Kernel(float* d_result, int width, int height)
{
	int row = (blockIdx.y << BLOCK_LOG_DIM) + threadIdx.y;
	int col = (blockIdx.x << BLOCK_LOG_DIM) + threadIdx.x;
	if(col < width && row < height)
	{
		int index = row * width  + col;
		float v = tex1Dfetch(texData, index);
		d_result[index] = (col == 0 || row == 0 || col == width -1 || row == height -1)?
			0.5 : saturate(0.5+20.0*v);
	}
}
///
void ProgramCU::DisplayConvertDOG(CuTexImage* dog, CuTexImage* out)
{
	if(out->_cuData == NULL) return;
	int width = dog->GetImgWidth(), height = dog ->GetImgHeight();
	dog->BindTexture(texData);
	dim3 grid((width + BLOCK_DIM - 1)/ BLOCK_DIM,  (height + BLOCK_DIM - 1)/BLOCK_DIM);
	dim3 block(BLOCK_DIM, BLOCK_DIM);
	ConvertDOG_Kernel<<<grid, block>>>((float*) out->_cuData, width, height);
	ProgramCU::CheckErrorCUDA("DisplayConvertDOG");
}

void __global__ ConvertGRD_Kernel(float* d_result, int width, int height)
{
	int row = (blockIdx.y << BLOCK_LOG_DIM) + threadIdx.y;
	int col = (blockIdx.x << BLOCK_LOG_DIM) + threadIdx.x;
	if(col < width && row < height)
	{
		int index = row * width  + col;
		float v = tex1Dfetch(texData, index << 1);
		d_result[index] = (col == 0 || row == 0 || col == width -1 || row == height -1)?
			0 : saturate(5 * v);

	}
}


void ProgramCU::DisplayConvertGRD(CuTexImage* got, CuTexImage* out)
{
	if(out->_cuData == NULL) return;
	int width = got->GetImgWidth(), height = got ->GetImgHeight();
	got->BindTexture(texData);
	dim3 grid((width + BLOCK_DIM - 1)/ BLOCK_DIM,  (height + BLOCK_DIM - 1)/BLOCK_DIM);
	dim3 block(BLOCK_DIM, BLOCK_DIM);
	ConvertGRD_Kernel<<<grid, block>>>((float*) out->_cuData, width, height);
	ProgramCU::CheckErrorCUDA("DisplayConvertGRD");
}

void __global__ ConvertKEY_Kernel(float4* d_result, int width, int height)
{

	int row = (blockIdx.y << BLOCK_LOG_DIM) + threadIdx.y;
	int col = (blockIdx.x << BLOCK_LOG_DIM) + threadIdx.x;
	if(col < width && row < height)
	{
		int index = row * width + col;
		float4 keyv = tex1Dfetch(texDataF4, index);
		int is_key = (keyv.x == 1.0f || keyv.x == -1.0f); 
		int inside = col > 0 && row > 0 && row < height -1 && col < width - 1;
		float v = inside? saturate(0.5 + 20 * tex1Dfetch(texData, index)) : 0.5;
		d_result[index] = is_key && inside ? 
			(keyv.x > 0? make_float4(1.0f, 0, 0, 1.0f) : make_float4(0.0f, 1.0f, 0.0f, 1.0f)): 
			make_float4(v, v, v, 1.0f) ;
	}
}
void ProgramCU::DisplayConvertKEY(CuTexImage* key, CuTexImage* dog, CuTexImage* out)
{
	if(out->_cuData == NULL) return;
	int width = key->GetImgWidth(), height = key ->GetImgHeight();
	dog->BindTexture(texData);
	key->BindTexture(texDataF4);
	dim3 grid((width + BLOCK_DIM - 1)/ BLOCK_DIM,  (height + BLOCK_DIM - 1)/BLOCK_DIM);
	dim3 block(BLOCK_DIM, BLOCK_DIM);
	ConvertKEY_Kernel<<<grid, block>>>((float4*) out->_cuData, width, height);
}


void __global__ DisplayKeyPoint_Kernel(float4 * d_result, int num)
{
	int idx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	if(idx >= num) return;
	float4 v = tex1Dfetch(texDataF4, idx);
	d_result[idx] = make_float4(v.x, v.y, 0, 1.0f);
}

void ProgramCU::DisplayKeyPoint(CuTexImage* ftex, CuTexImage* out)
{
	int num = ftex->GetImgWidth();
	int block_width = 64;
	dim3 grid((num + block_width -1) /block_width);
	dim3 block(block_width);
	ftex->BindTexture(texDataF4);
	DisplayKeyPoint_Kernel<<<grid, block>>>((float4*) out->_cuData, num);
	ProgramCU::CheckErrorCUDA("DisplayKeyPoint");
}

void __global__ DisplayKeyBox_Kernel(float4* d_result, int num)
{
	int idx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	if(idx >= num) return;
	int  kidx = idx / 10, vidx = idx - IMUL(kidx , 10);
	float4 v = tex1Dfetch(texDataF4, kidx);
	float sz = fabs(v.z * 3.0f);
	///////////////////////
	float s, c;	__sincosf(v.w, &s, &c);
	///////////////////////
	float dx = vidx == 0? 0 : ((vidx <= 4 || vidx >= 9)? sz : -sz);
	float dy = vidx <= 1? 0 : ((vidx <= 2 || vidx >= 7)? -sz : sz);
	float4 pos;
	pos.x = v.x + c * dx - s * dy;
	pos.y = v.y + c * dy + s * dx;
	pos.z = 0;	pos.w = 1.0f;
	d_result[idx]  = pos;
}

void ProgramCU::DisplayKeyBox(CuTexImage* ftex, CuTexImage* out)
{
	int len = ftex->GetImgWidth();
	int block_width = 32;
	dim3 grid((len * 10 + block_width -1) / block_width);
	dim3 block(block_width);
	ftex->BindTexture(texDataF4);
	DisplayKeyBox_Kernel<<<grid, block>>>((float4*) out->_cuData, len * 10);
}
///////////////////////////////////////////////////////////////////
inline void CuTexImage:: BindTexture(textureReference& texRef)
{
	cudaBindTexture(NULL, &texRef, _cuData, &texRef.channelDesc, _numBytes);
}

inline void CuTexImage::BindTexture2D(textureReference& texRef)
{
#if defined(SIFTGPU_ENABLE_LINEAR_TEX2D) 
	cudaBindTexture2D(0, &texRef, _cuData, &texRef.channelDesc, _imgWidth, _imgHeight, _imgWidth* _numChannel* sizeof(float));
#else
	cudaChannelFormatDesc desc;
	cudaGetChannelDesc(&desc, _cuData2D);
	cudaBindTextureToArray(&texRef, _cuData2D, &desc);
#endif
}

int ProgramCU::CheckCudaDevice(int device)
{
	int count = 0, device_used; 
	if(cudaGetDeviceCount(&count) != cudaSuccess  || count <= 0)
	{
		ProgramCU::CheckErrorCUDA("CheckCudaDevice");
		return 0;
	}else if(count == 1)
	{
		cudaDeviceProp deviceProp;
		if ( cudaGetDeviceProperties(&deviceProp, 0) != cudaSuccess  ||
			(deviceProp.major == 9999 && deviceProp.minor == 9999))
		{
			fprintf(stderr, "CheckCudaDevice: no device supporting CUDA.\n");
			return 0;
		}else
		{
			GlobalUtil::_MemCapGPU = deviceProp.totalGlobalMem / 1024;
			GlobalUtil::_texMaxDimGL = 132768;
			if(GlobalUtil::_verbose) 
				fprintf(stdout, "NOTE: changing maximum texture dimension to %d\n", GlobalUtil::_texMaxDimGL);

		}
	}
	if(device >0 && device < count)  
	{
		cudaSetDevice(device);
		CheckErrorCUDA("cudaSetDevice\n"); 
	}
	cudaGetDevice(&device_used);
	if(device != device_used) 
		fprintf(stderr,  "\nERROR:   Cannot set device to %d\n"
		"\nWARNING: Use # %d device instead (out of %d)\n", device, device_used, count);
	return 1;
}

////////////////////////////////////////////////////////////////////////////////////////
// siftmatch funtions
//////////////////////////////////////////////////////////////////////////////////////////

#define MULT_TBLOCK_DIMX 128
#define MULT_TBLOCK_DIMY 1
#define MULT_BLOCK_DIMX (MULT_TBLOCK_DIMX)
#define MULT_BLOCK_DIMY (8 * MULT_TBLOCK_DIMY)


texture<uint4, 1, cudaReadModeElementType> texDes1;
texture<uint4, 1, cudaReadModeElementType> texDes2;


//dim grid(num2/128,num1/8)
//dim block(128,1)
void __global__ MultiplyDescriptor_Kernel(int* d_result, int num1, int num2, int3* d_temp)
{
	//MULT_BLOCK_DIMY : 8         MULT_BLOCK_DIMX : 128
	int idx01 = (blockIdx.y  * MULT_BLOCK_DIMY),  //0~num1(0,8,16,24,32...)
		idx02 = (blockIdx.x  * MULT_BLOCK_DIMX);  //0~num2
	int idx1 = idx01 + threadIdx.y, //idx1 = idx01
		idx2 = idx02 + threadIdx.x;//col线程的列索引
	__shared__ int data1[17 * 2 * MULT_BLOCK_DIMY];  //每个线程块共享内存272，共享内存是一维的！ 线程块大小：（128,1）
	int read_idx1 = idx01 * 8 +  threadIdx.x,//用到了共享内存
		read_idx2 = idx2 * 8;//没用到共享内存
	int col4 = threadIdx.x & 0x3, //得到0,1,2,3
		row4 = threadIdx.x >> 2; //threadIdx.x/4
	int cache_idx1 = IMUL(row4, 17) + (col4 << 2);//row4*17+col4*4

	///////////////////////////////////////////////////////////////
	//Load feature descriptors
	///////////////////////////////////////////////////////////////
#if MULT_BLOCK_DIMY == 16
	uint4 v = tex1Dfetch(texDes1, read_idx1);
	data1[cache_idx1]   = v.x;	data1[cache_idx1+1] = v.y;
	data1[cache_idx1+2] = v.z;	data1[cache_idx1+3] = v.w;
#elif MULT_BLOCK_DIMY == 8
	if(threadIdx.x < 64) //threadIdx.x = 64时！！！cache_idx1为272！！！
	{
		uint4 v = tex1Dfetch(texDes1, read_idx1); //num1的索引
		data1[cache_idx1]   = v.x;		data1[cache_idx1+1] = v.y;
		data1[cache_idx1+2] = v.z;		data1[cache_idx1+3] = v.w;
	}
#else
#error
#endif
	__syncthreads();

	///
	if(idx2 >= num2) return;
	///////////////////////////////////////////////////////////////////////////
	//compare descriptors

	int results[MULT_BLOCK_DIMY]; //8个result
#pragma unroll
	for(int i = 0; i < MULT_BLOCK_DIMY; ++i) results[i] = 0;

#pragma unroll
	for(int i = 0; i < 8; ++i)  //8
	{
		uint4 v = tex1Dfetch(texDes2, read_idx2 + i);
		unsigned char* p2 = (unsigned char*)(&v);  //取出int包含了16个char
#pragma unroll
		for(int k = 0; k < MULT_BLOCK_DIMY; ++k)  //8
		{
			unsigned char* p1 = (unsigned char*) (data1 + k * 34 + i *  4 + (i/4)); //i/4是余数，因为每隔4个就加5
			results[k] += 	 ( IMUL(p1[0], p2[0])	+ IMUL(p1[1], p2[1])  
				+ IMUL(p1[2], p2[2])  	+ IMUL(p1[3], p2[3])  
				+ IMUL(p1[4], p2[4])  	+ IMUL(p1[5], p2[5])  
				+ IMUL(p1[6], p2[6])  	+ IMUL(p1[7], p2[7])  
				+ IMUL(p1[8], p2[8])  	+ IMUL(p1[9], p2[9])  
				+ IMUL(p1[10], p2[10])	+ IMUL(p1[11], p2[11])
				+ IMUL(p1[12], p2[12])	+ IMUL(p1[13], p2[13])
				+ IMUL(p1[14], p2[14])	+ IMUL(p1[15], p2[15]));

				//		results[k] += 	 ( IMUL(p1[0]-p2[0], p1[0]-p2[0])	+ IMUL(p1[1]-p2[1], p1[1]-p2[1])  
				//+ IMUL(p1[2]-p2[2], p1[2]-p2[2])  	+ IMUL(p1[3]-p2[3],p1[3]-p2[3] )  
				//+ IMUL(p1[4]-p2[4], p1[4]-p2[4])  	+ IMUL(p1[5]-p2[5], p1[5]-p2[5])  
				//+ IMUL(p1[6]-p2[6], p1[6]-p2[6])  	+ IMUL(p1[7]-p2[7], p1[7]-p2[7])  
				//+ IMUL(p1[8]-p2[8], p1[8]-p2[8])  	+ IMUL(p1[9]-p2[9], p1[9]-p2[9])  
				//+ IMUL(p1[10]-p2[10],p1[10]-p2[10])	+ IMUL(p1[11]-p2[11], p1[11]-p2[11])
				//+ IMUL(p1[12]-p2[12], p1[12]-p2[12])	+ IMUL(p1[13] -p2[13],p1[13] -p2[13])
				//+ IMUL(p1[14]-p2[14],p1[14]-p2[14] )	+ IMUL(p1[15]-p2[15], p1[15]-p2[15]));
		}
	}

	int dst_idx = IMUL(idx1, num2)  + idx2;  //(8*threadIdx.y*num2+idx2)
	if(d_temp)
	{
		int3 cmp_result = make_int3(0, -1, 0);  //8个result里面(最大距离，位置(num1上的)，次大值)

#pragma unroll
		for(int i = 0; i < MULT_BLOCK_DIMY; ++i)  //8
		{
			if(idx1 + i < num1)//i:0~8+idx1
			{
				cmp_result = results[i] > cmp_result.x? 	make_int3(results[i], idx1 + i, cmp_result.x) : 	make_int3(cmp_result.x, cmp_result.y, max(cmp_result.z, results[i]));
				//(i, num2)由于是共性内存，所以用了八个
				d_result[dst_idx + IMUL(i, num2)] = results[i];
			}
		}
		d_temp[ IMUL(blockIdx.y, num2) + idx2] = cmp_result; 
	}
	else
	{
#pragma unroll
		for(int i = 0; i < MULT_BLOCK_DIMY; ++i)//8
		{
			if(idx1 + i < num1) 
				d_result[dst_idx + IMUL(i, num2)] = results[i];
		}
	}

}


void ProgramCU::MultiplyDescriptor(CuTexImage* des1, CuTexImage* des2, CuTexImage* texDot, CuTexImage* texCRT)
{
	int num1 = des1->GetImgWidth() / 8;	//1067  num*8  4通道 float4
	int num2 = des2->GetImgWidth() / 8;   //728
	dim3 grid(	(num2 + MULT_BLOCK_DIMX - 1)/ MULT_BLOCK_DIMX,
		(num1 + MULT_BLOCK_DIMY - 1)/MULT_BLOCK_DIMY);// (num2/128,num1/8)
	dim3 block(MULT_TBLOCK_DIMX, MULT_TBLOCK_DIMY);//(128,1)
	//使用的线程的大小为(num1/8)*num2

	texDot->InitTexture( num2,num1);// (num2*num1)*4  float4

	if(texCRT)        //(num2 * num1/8 ) *4 float4
		texCRT->InitTexture(num2, (num1 + MULT_BLOCK_DIMY - 1)/MULT_BLOCK_DIMY, 32);
	des1->BindTexture(texDes1);//
	des2->BindTexture(texDes2);//


	//输入：CRT或NULL   texDes1和texDes2
	//输出：texDot

		GlobalUtil::StartTimer("竖直");
	MultiplyDescriptor_Kernel<<<grid, block>>>((int*)texDot->_cuData, num1, num2, 
		(texCRT? (int3*)texCRT->_cuData : NULL));
				cudaThreadSynchronize();			
				GlobalUtil::StopTimer();

		float _timing1 = GlobalUtil::GetElapsedTime();

				cudaThreadSynchronize();			


	ProgramCU::CheckErrorCUDA("MultiplyDescriptor");
}

texture<float, 1, cudaReadModeElementType> texLoc1;
texture<float2, 1, cudaReadModeElementType> texLoc2;
struct Matrix33{float mat[3][3];};


 //(num2/128，num1/8)
 //(128,8)
void __global__ MultiplyDescriptorG_Kernel(int* d_result, int num1, int num2, int3* d_temp,
	Matrix33 H, float hdistmax, Matrix33 F, float fdistmax)
{
	int idx01 = (blockIdx.y  * MULT_BLOCK_DIMY);	//8  （0,8,16,24,32...）
	int idx02 = (blockIdx.x  * MULT_BLOCK_DIMX);//128 (0,128,256...)

	int idx1 = idx01 + threadIdx.y;//行索引
	int idx2 = idx02 + threadIdx.x;//列索引
	__shared__ int data1[17 * 2 * MULT_BLOCK_DIMY];//272
	__shared__ float loc1[MULT_BLOCK_DIMY * 2];  //16
	int read_idx1 = idx01 * 8 +  threadIdx.x ; //8个代表一个描述子，
	int read_idx2 = idx2 * 8;  //8个代表一个描述子，
	int col4 = threadIdx.x & 0x3, row4 = threadIdx.x >> 2;
	int cache_idx1 = IMUL(row4, 17) + (col4 << 2);
#if MULT_BLOCK_DIMY == 16   //不执行
	uint4 v = tex1Dfetch(texDes1, read_idx1);
	data1[cache_idx1]   = v.x;
	data1[cache_idx1+1] = v.y;
	data1[cache_idx1+2] = v.z;
	data1[cache_idx1+3] = v.w; 
#elif MULT_BLOCK_DIMY == 8
	if(threadIdx.x < 64)  //小于64刚好272
	{
		uint4 v = tex1Dfetch(texDes1, read_idx1);
		data1[cache_idx1]   = v.x;
		data1[cache_idx1+1] = v.y;
		data1[cache_idx1+2] = v.z;
		data1[cache_idx1+3] = v.w;
	}
#else
#error
#endif
	__syncthreads();
	if(threadIdx.x < MULT_BLOCK_DIMY * 2) //小于16
	{
		// （0,8,16,24,32...）->（0,16,32,48,64...）+16
		loc1[threadIdx.x] = tex1Dfetch(texLoc1, 2 * idx01 + threadIdx.x);  //一个在共享内存里
	}
	__syncthreads();
	if(idx2 >= num2) return;
	int results[MULT_BLOCK_DIMY];
	/////////////////////////////////////////////////////////////////////////////////////////////
	//geometric verification    几何验证
	/////////////////////////////////////////////////////////////////////////////////////////////
	int good_count = 0;
	float2 loc2 = tex1Dfetch(texLoc2, idx2);  //一个在纹理内存里，负责定位   
	//idx2=
	//每个特征点对应
#pragma unroll
	for(int i = 0; i < MULT_BLOCK_DIMY; ++i)//8
	{
		if(idx1 + i < num1)
		{
			float* loci = loc1 + i * 2;
			float locx = loci[0], locy = loci[1];
			//homography
			float x[3], diff[2];
			x[0] = H.mat[0][0] * locx + H.mat[0][1] * locy + H.mat[0][2];
			x[1] = H.mat[1][0] * locx + H.mat[1][1] * locy + H.mat[1][2];
			x[2] = H.mat[2][0] * locx + H.mat[2][1] * locy + H.mat[2][2];
			diff[0] = fabs(FDIV(x[0], x[2]) - loc2.x);
			diff[1] = fabs(FDIV(x[1], x[2]) - loc2.y);
			if(diff[0] < hdistmax && diff[1] < hdistmax)
			{
				//check fundamental matrix 检查基础矩阵
				float fx1[3], ftx2[3], x2fx1, se; 
				fx1[0] = F.mat[0][0] * locx + F.mat[0][1] * locy + F.mat[0][2];
				fx1[1] = F.mat[1][0] * locx + F.mat[1][1] * locy + F.mat[1][2];
				fx1[2] = F.mat[2][0] * locx + F.mat[2][1] * locy + F.mat[2][2];

				ftx2[0] = F.mat[0][0] * loc2.x + F.mat[1][0] * loc2.y + F.mat[2][0];
				ftx2[1] = F.mat[0][1] * loc2.x + F.mat[1][1] * loc2.y + F.mat[2][1];
				//ftx2[2] = F.mat[0][2] * loc2.x + F.mat[1][2] * loc2.y + F.mat[2][2];

				x2fx1 = loc2.x * fx1[0]  + loc2.y * fx1[1] + fx1[2];
				se = FDIV(x2fx1 * x2fx1, fx1[0] * fx1[0] + fx1[1] * fx1[1] + ftx2[0] * ftx2[0] + ftx2[1] * ftx2[1]);
				results[i] = se < fdistmax? 0: -262144;
			}else
			{
				results[i] = -262144;
			}
		}else
		{
			results[i] = -262144;
		}
		good_count += (results[i] >=0);
	}
	/////////////////////////////////////////////////////////////////////////////////////////////
	///compare feature descriptors anyway 无论如何都要进行特征描述子生成，跟双向匹配没有任何区别！！！
	/////////////////////////////////////////////////////////////////////////////////////////////
	if(good_count > 0)
	{
#pragma unroll
		for(int i = 0; i < 8; ++i)
		{
			uint4 v = tex1Dfetch(texDes2, read_idx2 + i);
			unsigned char* p2 = (unsigned char*)(&v);
#pragma unroll
			for(int k = 0; k < MULT_BLOCK_DIMY; ++k)
			{
				unsigned char* p1 = (unsigned char*) (data1 + k * 34 + i *  4 + (i/4));
				results[k] += 	 ( IMUL(p1[0], p2[0])	+ IMUL(p1[1], p2[1])  
					+ IMUL(p1[2], p2[2])  	+ IMUL(p1[3], p2[3])  
					+ IMUL(p1[4], p2[4])  	+ IMUL(p1[5], p2[5])  
					+ IMUL(p1[6], p2[6])  	+ IMUL(p1[7], p2[7])  
					+ IMUL(p1[8], p2[8])  	+ IMUL(p1[9], p2[9])  
					+ IMUL(p1[10], p2[10])	+ IMUL(p1[11], p2[11])
					+ IMUL(p1[12], p2[12])	+ IMUL(p1[13], p2[13])
					+ IMUL(p1[14], p2[14])	+ IMUL(p1[15], p2[15]));
			}
		}
	}
	int dst_idx = IMUL(idx1, num2)  + idx2;
	if(d_temp)
	{
		int3 cmp_result = make_int3(0, -1, 0);
#pragma unroll
		for(int i= 0; i < MULT_BLOCK_DIMY; ++i)
		{
			if(idx1 + i < num1)
			{
				cmp_result = results[i] > cmp_result.x? 
					make_int3(results[i], idx1 + i, cmp_result.x) : 
				make_int3(cmp_result.x, cmp_result.y, max(cmp_result.z, results[i]));
				d_result[dst_idx + IMUL(i, num2)] = max(results[i], 0);
			}else
			{
				break;
			}
		}
		d_temp[ IMUL(blockIdx.y, num2) + idx2] = cmp_result; 
	}else
	{
#pragma unroll
		for(int i = 0; i < MULT_BLOCK_DIMY; ++i)
		{
			if(idx1 + i < num1) d_result[dst_idx + IMUL(i, num2)] = max(results[i], 0);
			else break;
		}
	}

}



void ProgramCU::MultiplyDescriptorG(CuTexImage* des1, CuTexImage* des2,
	CuTexImage* loc1, CuTexImage* loc2, CuTexImage* texDot, CuTexImage* texCRT,
	float H[3][3], float hdistmax, float F[3][3], float fdistmax)
{
	int num1 = des1->GetImgWidth() / 8;	
	int num2 = des2->GetImgWidth() / 8;
	Matrix33 MatF, MatH;
	//copy the matrix
	memcpy(MatF.mat, F, 9 * sizeof(float));
	memcpy(MatH.mat, H, 9 * sizeof(float));
	//thread blocks
	dim3 grid(	(num2 + MULT_BLOCK_DIMX - 1)/ MULT_BLOCK_DIMX,   //(num2/128，num1/8)
		(num1 + MULT_BLOCK_DIMY - 1)/MULT_BLOCK_DIMY);
	dim3 block(MULT_TBLOCK_DIMX, MULT_TBLOCK_DIMY);  //(128,8)
	//intermediate results 中间结果：双向匹配
	texDot->InitTexture( num2,num1);
	if(texCRT) texCRT->InitTexture( num2, (num1 + MULT_BLOCK_DIMY - 1)/MULT_BLOCK_DIMY, 3);
	loc1->BindTexture(texLoc1);	
	loc2->BindTexture(texLoc2);
	des1->BindTexture(texDes1);	//影像1
	des2->BindTexture(texDes2);//影像2
	MultiplyDescriptorG_Kernel<<<grid, block>>>((int*)texDot->_cuData, num1, num2, 
		(texCRT? (int3*)texCRT->_cuData : NULL),
		MatH, hdistmax, MatF, fdistmax);
}


texture<int,  1, cudaReadModeElementType> texDOT;

#define ROWMATCH_BLOCK_WIDTH 32
#define ROWMATCH_BLOCK_HEIGHT 1

void __global__  RowMatch_Kernel(int*d_dot, int* d_result, int num2, float distmax, float ratiomax)
{
#if ROWMATCH_BLOCK_HEIGHT == 1
	__shared__ int dotmax[ROWMATCH_BLOCK_WIDTH]; //32
	__shared__ int dotnxt[ROWMATCH_BLOCK_WIDTH];  //32
	__shared__ int dotidx[ROWMATCH_BLOCK_WIDTH];   //32
	int	row = blockIdx.y;
#else  //不使用
	__shared__ int x_dotmax[ROWMATCH_BLOCK_HEIGHT][ROWMATCH_BLOCK_WIDTH];   
	__shared__ int x_dotnxt[ROWMATCH_BLOCK_HEIGHT][ROWMATCH_BLOCK_WIDTH];
	__shared__ int x_dotidx[ROWMATCH_BLOCK_HEIGHT][ROWMATCH_BLOCK_WIDTH];
	int*	dotmax = x_dotmax[threadIdx.y];
	int*	dotnxt = x_dotnxt[threadIdx.y];
	int*	dotidx = x_dotidx[threadIdx.y];
	int row = IMUL(blockIdx.y, ROWMATCH_BLOCK_HEIGHT) + threadIdx.y;
#endif

	int base_address = IMUL(row , num2);  //dot层的索引！！！
	int t_dotmax = 0, t_dotnxt = 0, t_dotidx = -1;//最大值，次大值，索引
	for(int i = 0; i < num2; i += ROWMATCH_BLOCK_WIDTH)//步长32     说明threadIdx使用了num2/32次
	{
		if(threadIdx.x + i < num2)
		{
			int v = tex1Dfetch(texDOT, base_address + threadIdx.x + i);//d_dot[base_address + threadIdx.x + i];//
			bool test = v > t_dotmax;  //最大值
			t_dotnxt = test? t_dotmax : max(t_dotnxt, v);//次大值
			t_dotidx = test? (threadIdx.x + i) : t_dotidx; //只记录32个共享内存里最大值的索引！！！
			t_dotmax = test? v: t_dotmax;
		}
		__syncthreads();//在一个线程块之行结束后，t_dotmax已经被赋值，这个时候等待其它线程块求最大值
		//所有线程块都计算完之后才同步,，然后重复利用线程块，进行其他区域的判定！
	}
	//已经求得所有区域的最值！！！
	dotmax[threadIdx.x] = t_dotmax;
	dotnxt[threadIdx.x] = t_dotnxt;
	dotidx[threadIdx.x] = t_dotidx;
	__syncthreads();

#pragma unroll
	for(int step = ROWMATCH_BLOCK_WIDTH/2; step >0; step /= 2) //归约算法，每次分为两部分
	{
		if(threadIdx.x < step)
		{
			int v1 = dotmax[threadIdx.x], v2 = dotmax[threadIdx.x + step];
			bool test =  v2 > v1;
			dotnxt[threadIdx.x] = test? max(v1, dotnxt[threadIdx.x + step]) :max(dotnxt[threadIdx.x], v2);  //次大值和v1v2中较小的相比较
			dotidx[threadIdx.x] = test? dotidx[threadIdx.x + step] : dotidx[threadIdx.x];
			dotmax[threadIdx.x] = test? v2 : v1;
		}
		__syncthreads();
	}
	if(threadIdx.x == 0)
	{
		float dist =  acos(min(dotmax[0] * 0.000003814697265625f, 1.0));   //最大值，但是acos是减函数，所以得到的较小值
		float distn = acos(min(dotnxt[0] * 0.000003814697265625f, 1.0));   //次大值，但是acos是减函数，所以得到反而是较大值
		//float ratio = dist / distn;
		d_result[row] = (dist < distmax) && (dist < distn * ratiomax) ? dotidx[0] : -1;//
	}

}


void ProgramCU::GetRowMatch(CuTexImage* texDot, CuTexImage* texMatch, float distmax, float ratiomax)
{
	int num1 = texDot->GetImgHeight();//1068
	int num2 = texDot->GetImgWidth();//731
	dim3 grid(1, num1/ROWMATCH_BLOCK_HEIGHT);//dim3 grid(1,num1)
	dim3 block(ROWMATCH_BLOCK_WIDTH, ROWMATCH_BLOCK_HEIGHT);//dim3 block(32,1)
	texDot->BindTexture(texDOT);


				GlobalUtil::StartTimer("竖直");

	RowMatch_Kernel<<<grid, block>>>((int*)texDot->_cuData,
		(int*)texMatch->_cuData, num2, distmax, ratiomax);
		cudaThreadSynchronize();
				GlobalUtil::StopTimer();
		float _timing1 = GlobalUtil::GetElapsedTime();

}

#define COLMATCH_BLOCK_WIDTH 32

//texture<int3,  1, cudaReadModeElementType> texCT;

void __global__  ColMatch_Kernel(int3*d_crt, int* d_result, int height, int num2, float distmax, float ratiomax)
{
	int col = COLMATCH_BLOCK_WIDTH * blockIdx.x + threadIdx.x; //列
	if(col >= num2) return;
	int3 result = d_crt[col];//tex1Dfetch(texCT, col);
	int read_idx = col + num2;
	for(int i = 1; i < height; ++i, read_idx += num2)
	{
		int3 temp = d_crt[read_idx];//tex1Dfetch(texCT, read_idx);
		result = result.x < temp.x?   //取较大值
			make_int3(temp.x, temp.y, max(result.x, temp.z)) :
		make_int3(result.x, result.y, max(result.z, temp.x));
	}

	float dist =  acos(min(result.x * 0.000003814697265625f, 1.0));
	float distn = acos(min(result.z * 0.000003814697265625f, 1.0));
	//float ratio = dist / distn;
	d_result[col] = (dist < distmax) && (dist < distn * ratiomax) ? result.y : -1;//

}

void ProgramCU::GetColMatch(CuTexImage* texCRT, CuTexImage* texMatch, float distmax, float ratiomax)
{
	int height = texCRT->GetImgHeight();
	int num2 = texCRT->GetImgWidth();
	//texCRT->BindTexture(texCT);
	dim3 grid((num2 + COLMATCH_BLOCK_WIDTH -1) / COLMATCH_BLOCK_WIDTH);//num2/32
	dim3 block(COLMATCH_BLOCK_WIDTH);  //32*1
	ColMatch_Kernel<<<grid, block>>>((int3*)texCRT->_cuData, (int*) texMatch->_cuData, height, num2, distmax, ratiomax);
		cudaThreadSynchronize();
}

#endif
